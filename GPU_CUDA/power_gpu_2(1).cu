#include "hip/hip_runtime.h"
// the subroutine for GPU code can be found in several separated text file from the Brightspace. 
// You can add these subroutines to this main code.
////////////////////////////////////////////


#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"


const int BLOCK_SIZE = 32;  // number of threads per block

float* h_MatA = NULL;
float* d_MatA = NULL;

// Output Array
float* h_Lamda = NULL;
float* d_Lamda = NULL;
float* h_VecV = NULL;
float* d_VecV = NULL;
float* h_VecW = NULL;
float* d_VecW = NULL;
float* h_NormW = NULL;
float* d_NormW = NULL;

// Variables to change
int GlobalSize = 2000;         // this is the dimension of the matrix, GlobalSize*GlobalSize
int BlockSize = BLOCK_SIZE;            // number of threads in each block
const float EPS = 0.000005;    // tolerence of the error
int max_iteration = 100;       // the maximum iteration steps


// Functions
void Cleanup(void);
void InitOne(float*, int);
void UploadArray(float*, int);
float CPUReduce(float*, int);
void  ParseArguments(int, char**);
void checkCardVersion(void);
void checkCudaError(hipError_t, const char[], int);
void matrixWriter(float * , int , int , const char[]);

// Kernels
__global__ void Av_Product(float* g_MatA, float* g_VecV, float* g_VecW, int N);
__global__ void FindNormW(float* g_VecW, float * g_NormW, int N);
__global__ void NormalizeW(float* g_VecW, float * g_NormW, float* g_VecV, int N); 
__global__ void ComputeLamda( float* g_VecV,float* g_VecW, float * g_Lamda,int N);


void CPU_AvProduct()
{
	int N = GlobalSize;
	int matIndex =0;
    for(int i=0;i<N;i++)
	{
		h_VecW[i] = 0;
		for(int j=0;j<N;j++)
		{
			matIndex = i*N + j;
			h_VecW[i] += h_MatA[matIndex] * h_VecV[j];
			
		}
	}
}

void CPU_NormalizeW()
{
	int N = GlobalSize;
	float normW=0;
	for(int i=0;i<N;i++)
		normW += h_VecW[i] * h_VecW[i];

	normW = sqrt(normW);

    //printf("NormW-CPU: %f\n", normW);    
	for(int i=0;i<N;i++)
		h_VecV[i] = h_VecW[i]/normW;
}

float CPU_ComputeLamda()
{
	int N = GlobalSize;
	float lamda =0;
	for(int i=0;i<N;i++)
		lamda += h_VecV[i] * h_VecW[i];
	
	return lamda;
}

void RunCPUPowerMethod()
{
	printf("*************************************\n");
	float oldLamda =0;
	float lamda=0;
    int N = GlobalSize;


    //matrixWriter(h_MatA, N, N, "matA.mat");
    //matrixWriter(h_VecV, 1, N, "matV.mat"); 
	//AvProduct
	CPU_AvProduct();

    //matrixWriter(h_VecW, 1, N, "matW.mat");
	
	//power loop
    int i;
	for (i=0;i<max_iteration;i++)
	{
		CPU_NormalizeW();
		CPU_AvProduct();
		lamda= CPU_ComputeLamda();
		//printf("CPU lamda at %d: %f \n", i, lamda);
		// If residual is lass than epsilon break
		if(abs(oldLamda - lamda) < EPS)
			break;
		oldLamda = lamda;	
	
	}
    printf("CPU lamda at %d: %f \n", i, lamda);
	printf("*************************************\n");
	
}


// Host code
int main(int argc, char** argv)
{

    struct timespec t_start,t_end;
    double runtime;
    ParseArguments(argc, argv);
		
    int N = GlobalSize;
    printf("Matrix size %d X %d || Blocksize: %i \n", N, N, BlockSize);
    size_t vec_size = N * sizeof(float);
    size_t mat_size = N * N * sizeof(float);
    size_t norm_size = sizeof(float);
  
    // Allocate normalized value in host memory
    h_NormW = (float*)malloc(norm_size);
    // Allocate input matrix in host memory
    h_MatA = (float*)malloc(mat_size);
    // Allocate initial vector V in host memory
    h_VecV = (float*)malloc(vec_size);
    // Allocate W vector for computations
    h_VecW = (float*)malloc(vec_size);
    // Allocate lamda value in host memory
    h_Lamda = (float *)malloc(norm_size);


    // Initialize input matrix
    UploadArray(h_MatA, N);
    InitOne(h_VecV,N);

    printf("Power method in CPU starts\n");	   
    clock_gettime(CLOCK_REALTIME,&t_start);
    RunCPUPowerMethod();   // the lamda is already solved here
    clock_gettime(CLOCK_REALTIME,&t_end);
    runtime = (t_end.tv_sec - t_start.tv_sec) + 1e-9*(t_end.tv_nsec - t_start.tv_nsec);
    printf("CPU: run time = %f secs.\n",runtime);
    printf("Power method in CPU is finished\n");
    
    
    /////////////////////////////////////////////////
    // This is the starting points of GPU
    printf("Power method in GPU starts\n");
    checkCardVersion();

    // Initialize input matrix
    InitOne(h_VecV,N);
    
    clock_gettime(CLOCK_REALTIME,&t_start);  // Here I start to count

    // Set the kernel arguments
    int threadsPerBlock = BlockSize;   
    int sharedMemSize = threadsPerBlock * threadsPerBlock * sizeof(float); // in per block, the memory is shared   
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t cuda_err;
    // Allocate matrix and vectors in device memory
    cuda_err = hipMalloc((void**)&d_MatA, mat_size);
    checkCudaError(cuda_err, "Error Allocating Matrix", 1);
    cuda_err = hipMalloc((void**)&d_VecV, vec_size);
    checkCudaError(cuda_err, "Error Allocating Vector", 1);
    cuda_err = hipMalloc((void**)&d_VecW, vec_size); // This vector is only used by the device
    checkCudaError(cuda_err, "Error Allocating Normal Vector", 1);
    cuda_err = hipMalloc((void**)&d_NormW, norm_size); 
    checkCudaError(cuda_err, "Error Value for Normallised Eigen Vector", 1);
    cuda_err = hipMalloc((void**)&d_Lamda, norm_size);
    checkCudaError(cuda_err, "Error Allocating Lamda", 1);


    cuda_err = hipMemset(d_VecW, 0, vec_size);
    checkCudaError(cuda_err, "Error Setting Vector Size", 1);

    //Copy from host memory to device memory
    cuda_err = hipMemcpy(d_MatA, h_MatA, mat_size, hipMemcpyHostToDevice);
    checkCudaError(cuda_err, "Error Copying host matrix to device", 1);
    cuda_err = hipMemcpy(d_VecV, h_VecV, vec_size, hipMemcpyHostToDevice);
    checkCudaError(cuda_err, "Error Copying host vector to device", 1);
	// cutilCheckError(cutStopTimer(timer_mem));
	  
   //PowerG method loops
    float OldLamda =0;
    //matrixWriter(h_MatA, N, N, "gpuMatA.mat");
    //matrixWriter(h_VecV, 1, N, "gpuMatV.mat"); 
    Av_Product<<<blocksPerGrid, threadsPerBlock>>>(d_MatA, d_VecV, d_VecW, N);
    cuda_err = hipGetLastError();
    checkCudaError(cuda_err, "Sync Error with Av_Product", 1);
    cuda_err = hipDeviceSynchronize();
    checkCudaError(cuda_err, "Async Error with Av_Product", 1);
    
    //cuda_err = hipMemcpy(h_VecW, d_VecW, vec_size, hipMemcpyDeviceToHost);
    //matrixWriter(h_VecW, 1, N, "gpuMatW.mat");

    int idx;
    for (idx = 0; idx < max_iteration; idx++) {

        cuda_err = hipMemset(d_NormW, 0, norm_size);
        FindNormW<<<blocksPerGrid, threadsPerBlock>>> (d_VecW, d_NormW, N);

        cuda_err = hipGetLastError();
        checkCudaError(cuda_err, "Sync Error with FindNormW", 1);
        cuda_err = hipDeviceSynchronize();
        checkCudaError(cuda_err, "Async Error with FindNormW", 1);  
        cuda_err = hipMemcpy(h_NormW, d_NormW, norm_size, hipMemcpyDeviceToHost);
        checkCudaError(cuda_err, "Error copying NormW to Host", 1);

        
        h_NormW[0] = sqrt(h_NormW[0]);
        //printf("NormW: %.4f\n", h_NormW[0]);
        cuda_err = hipMemcpy(d_NormW, h_NormW, norm_size, hipMemcpyHostToDevice);
        checkCudaError(cuda_err, "Error Setting new value of NormW on Device", 1);

        NormalizeW<<<blocksPerGrid, threadsPerBlock >>> (d_VecW, d_NormW, d_VecV, N);
        cuda_err = hipGetLastError();
        checkCudaError(cuda_err, "Sync Error with Normalize W", 1);
        cuda_err = hipDeviceSynchronize();
        checkCudaError(cuda_err, "Async Error with NormalizeW", 1);
        
        Av_Product<<<blocksPerGrid, threadsPerBlock>>>(d_MatA, d_VecV, d_VecW, N);
        cuda_err = hipGetLastError();
        checkCudaError(cuda_err, "Sync Error with Av_Product", 1);
        cuda_err = hipDeviceSynchronize();
        checkCudaError(cuda_err, "Async Error with Av_Product", 1);
        
        cuda_err = hipMemset(d_Lamda, 0, norm_size);
        checkCudaError(cuda_err, "Error Setting value of lamda to zero", 1);
        
        ComputeLamda<<<blocksPerGrid, threadsPerBlock>>>  (d_VecV, d_VecW, d_Lamda, N);
        cuda_err = hipGetLastError();
        checkCudaError(cuda_err, "Sync Error with Compute Lamda", 1);

        cuda_err = hipDeviceSynchronize();
        checkCudaError(cuda_err, "Async Error with Compute Lamda", 1);

        cuda_err = hipMemcpy(h_Lamda, d_Lamda, norm_size, hipMemcpyDeviceToHost);
        checkCudaError(cuda_err, "Error copying device lamda to host", 1);

        //printf("GPU lamda at %d: %f \n", idx, h_Lamda[0]);

        if(abs(OldLamda - h_Lamda[0]) < EPS)
			break;
        OldLamda = h_Lamda[0];
    }
    printf("GPU lamda at %d: %f \n", idx, h_Lamda[0]);
	
    // This part is the main code of the iteration process for the Power Method in GPU. 
    // Please finish this part based on the given code. Do not forget the command line 
    // hipDeviceSynchronize() after callig the function every time in CUDA to synchoronize the threads
    ////////////////////////////////////////////
    //   ///      //        //            //          //            //        //
    //                                                                        //
    //                                                                        //
    //                                                                        //
    //                                                                        //
    //                                                                        //
    //                                                                        //
    //                                                                        //
    //  ///   //    ///     //    //      //      //        //       //   //  //
    
    

    clock_gettime(CLOCK_REALTIME,&t_end);
    runtime = (t_end.tv_sec - t_start.tv_sec) + 1e-9*(t_end.tv_nsec - t_start.tv_nsec);
    printf("GPU: run time = %f secs.\n",runtime);
    // printf("Overall CPU Execution Time: %f (ms) \n", cutGetTimerValue(timer_CPU));

    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_MatA)
        hipFree(d_MatA);
    if (d_VecV)
        hipFree(d_VecV);
    if (d_VecW)
        hipFree(d_VecW);
	if (d_NormW)
		  hipFree(d_NormW);
    if (d_Lamda)
        hipFree(d_Lamda);
		
    // Free host memory
    if (h_MatA)
        free(h_MatA);
    if (h_VecV)
        free(h_VecV);
    if (h_VecW)
        free(h_VecW);
    if (h_NormW)
        free(h_NormW);
    if (h_Lamda)
        free(h_Lamda);
    
    exit(0);
}

// Allocates an array with zero value.
void InitOne(float* data, int n)
{
    for (int i = 0; i < n; i++)
        data[i] = 0;
	data[0]=1;
}

void UploadArray(float* data, int n)
{
   int total = n*n;
   int value=1;
    for (int i = 0; i < total; i++)
    {
    	data[i] = (int) (3*rand() % (int)(101));//1;//value;
	    value ++; if(value>n) value =1;
      // data[i] = 1;
    }
}

// Obtain program arguments
void ParseArguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) 
    {
        if (strcmp(argv[i], "--size") == 0 || strcmp(argv[i], "-size") == 0)
        {
            GlobalSize = atoi(argv[i+1]);
		    i = i + 1;
        }
        if (strcmp(argv[i], "--max_iteration") == 0 || strcmp(argv[i], "-max_iteration") == 0)
        {
            max_iteration = atoi(argv[i+1]);
		    i = i + 1;
        }
    }
}


void checkCardVersion()
{
   hipDeviceProp_t prop;
   
   hipGetDeviceProperties(&prop, 0);
   
   printf("This GPU has major architecture %d, minor %d \n",prop.major,prop.minor);
   if(prop.major < 2)
   {
      fprintf(stderr,"Need compute capability 2 or higher.\n");
      exit(1);
   }
}

/*****************************************************************************
This function finds the product of Matrix A and vector V
*****************************************************************************/

// ****************************************************************************************************************************************************/
// parallelization method for the Matrix-vector multiplication as follows:

// each thread handle a multiplication of each row of Matrix A and vector V;

// The share memory is limited for a block, instead of reading an entire row of matrix A or vector V from global memory to share memory,
// a square submatrix of A is shared by a block, the size of square submatrix is BLOCK_SIZE*BLOCK_SIZE; Thus, a for-loop is used to
// handle a multiplication of each row of Matrix A and vector V step by step. In eacg step, two subvectors with size BLOCK_SIZE is multiplied.
//*****************************************************************************************************************************************************/


__global__ void Av_Product(float* g_MatA, float* g_VecV, float* g_VecW, int N)
{
    unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = globalid; idx < N; idx+= stride) {
        float sum = 0.0;
        for (int jdx = 0; jdx < N; jdx ++) {
            int mat_index = idx* N + jdx;
            sum += g_VecV[jdx] * g_MatA[mat_index];
        }

        g_VecW[idx] = sum;
    }
}


__global__ void ComputeLamda( float* g_VecV, float* g_VecW, float * g_Lamda,int N)
{

  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  // For thread ids greater than data space
  float product;
  for(int idx = globalid; idx < N; idx += stride) {
     product = g_VecV[idx] * g_VecW[idx];
     atomicAdd(g_Lamda, product);
  }
}


__global__ void NormalizeW(float* g_VecW, float * g_NormW, float* g_VecV, int N)
{

  float normal = g_NormW[0];
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int idx = globalid; idx < N; idx += stride) {
      g_VecV[idx] = g_VecW[idx]/normal;
  }

}

/****************************************************
Normalizes vector W : W/norm(W)
****************************************************/
__global__ void FindNormW(float* g_VecW, float * g_NormW, int N)
{
  // shared memory size declared at kernel launch
  unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  float square_value;

  for (int idx = globalid; idx < N; idx += stride) {
      square_value = g_VecW[globalid];
      square_value = square_value * square_value;
      atomicAdd(g_NormW,square_value);
  }
}

void checkCudaError(hipError_t cuda_err, const char mesg[], int terminate) {
    bool isError = cuda_err != hipSuccess;
    if (isError) {
        printf("Reason for Error: %s\n", hipGetErrorString(cuda_err));
        printf("%s\n", mesg);
    }


    if (isError && terminate) {
        exit(1);
    }
}

void matrixWriter(float * matrix, int xdim, int ydim, const char filename[]) {
    FILE *f;
    if ((f = fopen(filename, "w")) == NULL) {
        printf("Failed to write file\n");
        return;
    }

    float value;
    int index;
    for (int idx = 0; idx < xdim; idx ++) {
        for (int jdx = 0; jdx < ydim; jdx ++ ){
            index = idx * xdim + jdx;
            value = matrix[index];
            if (jdx+1 != ydim) {
                fprintf(f, "%.2f,", value); 
            } else {
                fprintf(f, "%.2f\n", value);
            }
        }
    }

    fclose(f);
}
